#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>


rtBuffer<optix::float3, 2> result_buffer;

rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );
rtDeclareVariable(float, redValue, ,);
RT_PROGRAM void rayGeneration() {
	optix::float3 red = make_float3(0.0f);
	red.x = redValue;
	result_buffer[theLaunchIndex] = red;
}